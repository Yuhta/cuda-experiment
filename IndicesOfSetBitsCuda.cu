#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>

#include "IndicesOfSetBits.h"

void checkCuda(hipError_t result) {
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    abort();
  }
}

constexpr int kByteCount = 1 << 20;
constexpr int kRepsCount = 100;
constexpr int kBlockSize = 256;
constexpr int kBlockCount = kByteCount / (sizeof(uint32_t) * kBlockSize * 2);

__device__ uint32_t exclusiveScan(uint32_t* a) {
  int d = 1;
  while (d <= blockDim.x) {
    int i = (threadIdx.x + 1) * d * 2 - 1;
    if (i < 2 * blockDim.x) {
      a[i] += a[i - d];
    }
    __syncthreads();
    d *= 2;
  }
  uint32_t ans = 0;
  if (threadIdx.x == 0) {
    ans = a[2 * blockDim.x - 1];
    a[2 * blockDim.x - 1] = 0;
  }
  __syncthreads();
  d = blockDim.x;
  while (d > 0) {
    int i = (threadIdx.x + 1) * d * 2 - 1;
    if (i < 2 * blockDim.x) {
      auto tmp = a[i];
      a[i] += a[i - d];
      a[i - d] = tmp;
    }
    __syncthreads();
    d /= 2;
  }
  return ans;
}

__global__ void computePositions1(const uint32_t* bits, uint32_t* blockSums, uint32_t* positions) {
  __shared__ uint32_t pos[2 * kBlockSize];
  auto i = 2 * blockIdx.x * blockDim.x + threadIdx.x;
  pos[threadIdx.x] = __popc(bits[i]);
  pos[threadIdx.x + kBlockSize] = __popc(bits[i + kBlockSize]);
  __syncthreads();
  auto sum = exclusiveScan(pos);
  if (threadIdx.x == 0) {
    blockSums[blockIdx.x] = sum;
  }
  positions[i] = pos[threadIdx.x];
  positions[i + kBlockSize] = pos[threadIdx.x + kBlockSize];
}

__global__ void computePositions2(uint32_t* blockSums, int* outCount) {
  __shared__ uint32_t a[kBlockCount];
  auto i = threadIdx.x;
  a[i] = blockSums[i];
  a[i + blockDim.x] = blockSums[i + blockDim.x];
  __syncthreads();
  auto sum = exclusiveScan(a);
  if (threadIdx.x == 0) {
    *outCount = sum;
  }
  blockSums[i] = a[i];
  blockSums[i + blockDim.x] = a[i + blockDim.x];
}

template <bool kWriteGlobal>
__global__ void writeResult(const uint32_t* bits, const uint32_t* blockSums, const uint32_t* positions, int32_t* indices) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockSums[blockIdx.x / 2] + positions[i];
  auto w = bits[i];
  int32_t ans{};
  while (w) {
    if constexpr (kWriteGlobal) {
      indices[j++] = i * 32 + __ffs(w) - 1;
    } else {
      ans ^= i * 32 + __ffs(w) - 1;
    }
    w &= w - 1;
  }
  if constexpr (!kWriteGlobal) {
    if (i == 0) {
      indices[0] = ans;
    }
  }
}

struct IndicesOfSetBitsCuda {
  IndicesOfSetBitsCuda(const void* bits) {
    checkCuda(hipMalloc(&bits_, kByteCount));
    checkCuda(hipMalloc(&indices_, sizeof(int32_t) * 8 * kByteCount));
    checkCuda(hipMalloc(&blockSums_, sizeof(uint32_t) * kBlockCount));
    checkCuda(hipMalloc(&positions_, kByteCount));
    checkCuda(hipMalloc(&count_, sizeof(int)));
    checkCuda(hipMemcpy(bits_, bits, kByteCount, hipMemcpyHostToDevice));
    checkCuda(hipHostMalloc(&actual_, sizeof(int32_t) * 8 * kByteCount));
  }

  void run() {
    computePositions1<<<kBlockCount, kBlockSize>>>(bits_, blockSums_, positions_);
    // Invoke a separate kernel to synchronize different blocks.
    static_assert(kBlockCount % 2 == 0);
    computePositions2<<<1, kBlockCount / 2>>>(blockSums_, count_);
    writeResult<true><<<2*kBlockCount, kBlockSize>>>(bits_, blockSums_, positions_, indices_);
  }

  void validate(int expectedCount, const int32_t* expected) {
    int actualCount;
    checkCuda(hipMemcpy(&actualCount, count_, sizeof(int), hipMemcpyDeviceToHost));
    assert(actualCount == expectedCount);
    checkCuda(hipMemcpy(actual_, indices_, sizeof(int32_t) * actualCount, hipMemcpyDeviceToHost));
    for (int i = 0; i < actualCount; ++i) {
      assert(actual_[i] == expected[i]);
    }
  }

private:
  // Device memory
  uint32_t* bits_;
  int* count_;
  int32_t* indices_;
  uint32_t* blockSums_;
  uint32_t* positions_;

  // Host memory
  int32_t* actual_; // too large to put on stack
};

char bits[kByteCount];
int32_t expected[8 * kByteCount];

int main() {
  for (int i = 0; i < kByteCount; ++i) {
    char byte{};
    for (int j = 0; j < 8; ++j) {
      byte |= ((rand() % 100) < 50) << j;
    }
    bits[i] = byte;
  }

  hipEvent_t startEvent, stopEvent;
  float ms;
  checkCuda(hipEventCreate(&startEvent));
  checkCuda(hipEventCreate(&stopEvent));

  int expectedCount;
  indicesOfSetBits(reinterpret_cast<const uint64_t*>(bits), 8 * kByteCount, expectedCount, expected);
  checkCuda(hipEventRecord(startEvent));
  for (int i = 0; i < kRepsCount; ++i) {
    int n;
    indicesOfSetBits(reinterpret_cast<const uint64_t*>(bits), 8 * kByteCount, n, expected);
    assert(n == expectedCount);
  }
  checkCuda(hipEventRecord(stopEvent));
  checkCuda(hipEventSynchronize(stopEvent));
  checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
  printf("CPU Implementation: %.2f ms\n", ms);

  // checkCuda(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));
  IndicesOfSetBitsCuda runner(bits);
  runner.run();
  runner.validate(expectedCount, expected);
  checkCuda(hipEventRecord(startEvent));
  for (int i = 0; i < kRepsCount; ++i) {
    runner.run();
  }
  checkCuda(hipEventRecord(stopEvent));
  checkCuda(hipEventSynchronize(stopEvent));
  checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
  printf("GPU Implementation: %.2f ms\n", ms);

  return 0;
}
